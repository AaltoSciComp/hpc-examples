// Using CUDA device to calculate pi
#include <stdio.h>
#include <hip/hip_runtime.h>

#define NBIN 10000000  // Number of bins
#define NUM_BLOCK  30  // Number of thread blocks
#define NUM_THREAD  8  // Number of threads per block
int tid;
float pi = 0;

// Kernel that executes on the CUDA device
__global__ void cal_pi(float *sum, int nbin, float step, int nthreads, int nblocks) {
	int i;
	float x;
	int idx = blockIdx.x*blockDim.x+threadIdx.x;  // Sequential thread index across the blocks
	for (i=idx; i< nbin; i+=nthreads*nblocks) {
		x = (i+0.5)*step;
		sum[idx] += 4.0/(1.0+x*x);
	}
}

// Main routine that executes on the host
int main(void) {
	dim3 dimGrid(NUM_BLOCK,1,1);  // Grid dimensions
	dim3 dimBlock(NUM_THREAD,1,1);  // Block dimensions
	float *sumHost, *sumDev;  // Pointer to host & device arrays

	float step = 1.0/NBIN;  // Step size
	size_t size = NUM_BLOCK*NUM_THREAD*sizeof(float);  //Array memory size
	sumHost = (float *)malloc(size);  //  Allocate array on host
	hipMalloc((void **) &sumDev, size);  // Allocate array on device
	// Initialize array in device to 0
	hipMemset(sumDev, 0, size);
	// Do calculation on device
	cal_pi <<<dimGrid, dimBlock>>> (sumDev, NBIN, step, NUM_THREAD, NUM_BLOCK); // call CUDA kernel
	// Retrieve result from device and store it in host array
	hipMemcpy(sumHost, sumDev, size, hipMemcpyDeviceToHost);
	for(tid=0; tid<NUM_THREAD*NUM_BLOCK; tid++)
		pi += sumHost[tid];
	pi *= step;

	// Print results
	printf("PI = %f\n",pi);

	// Cleanup
	free(sumHost); 
	hipFree(sumDev);

	return 0;
}
