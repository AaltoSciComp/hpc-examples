#include "hip/hip_runtime.h"

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdint.h>
#include <stdio.h>

// Setup random number generator
__global__ void setup_rng(hiprandState *random_states,  uint64_t seed)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(seed, tid, 0, &random_states[tid]);
}

// Throw nthrows per thread
__global__ void throw_dart(hiprandState *random_states, int *nthrows, uint64_t *hits)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  float random_x, random_y;
  hiprandState random_state = random_states[tid];

  hits[tid] = 0;
  for (int i=0; i<nthrows[tid]; i++) {
    random_x = hiprand_uniform(&random_state);
    random_y = hiprand_uniform(&random_state);

    if ((random_x*random_x + random_y*random_y) < 1.0) {
      hits[tid] += 1;
    }
  }
}

int main(int argc, char **argv) {

  // pi init
  long N=10000000;
  if (argc > 1)
    sscanf(argv[1], "%ld", &N);
  printf("Calculating pi using %ld stochastic trials\n", N);

  // Initialize variables
  int count, device;

  int *nthrows, *nthrows_gpu;
  uint64_t seed = 5;
  uint64_t *hits, *hits_gpu;
  hiprandState* random_states;
  uint64_t total_hits;
  float pi;

  // Run 2048 blocks
  int blocks = 512;
  // Run 128 threads per block.
  int threads = 128;

  int batch_size = blocks * threads;

  // Select device
  hipGetDeviceCount(&count);
  hipGetDevice(&device);

  // Allocate memory
  hits = (uint64_t*) malloc(batch_size*sizeof(uint64_t));
  nthrows = (int *) malloc(batch_size*sizeof(int));
  hipMalloc(&hits_gpu, batch_size*sizeof(uint64_t));
  hipMalloc(&nthrows_gpu, batch_size*sizeof(int));
  hipMalloc(&random_states, batch_size*sizeof(hiprandState));

  // Calculate how many throws we want per thread
  for (int i=0; i<batch_size; i++) {
    nthrows[i] = N / batch_size;
    if (i < N % batch_size) {
      nthrows[i] += 1;
    }
  }

  // Copy throw number info to GPU VRAM
  hipMemcpy(nthrows_gpu, nthrows, batch_size*sizeof(int), hipMemcpyHostToDevice);

  // Initialize random number generator for each thread
  setup_rng<<<blocks, threads>>>(random_states, seed);

  // Throw darts
  throw_dart<<<blocks, threads>>>(random_states, nthrows_gpu, hits_gpu);

  // Copy hits to host RAM
  hipMemcpy(hits, hits_gpu, batch_size*sizeof(uint64_t), hipMemcpyDeviceToHost);

  // Calculate the total number of hits
  total_hits = 0;
  for (int i=0; i<batch_size; i++) {
    total_hits += hits[i];
  }

  // Calculate pi
  pi = (double) total_hits*4/N;
  printf("Throws: %lu/%lu Pi: %.10g\n", total_hits, N, pi);

  // Free memory
  free(hits);
  free(nthrows);
  hipFree(hits_gpu);
  hipFree(nthrows_gpu);
  hipFree(random_states);

  return (0);
}
